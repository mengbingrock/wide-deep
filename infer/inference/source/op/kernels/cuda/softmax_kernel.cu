#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>
#include "softmax_kernel.cuh"
namespace kernel {
__global__ void row_softmax_fp32(const float* in, int size) {
  const int tid = threadIdx.x;
  const int lane_id = tid % warpSize;
  const float* x = in;
  float* y = const_cast<float*>(in);

  float max_val = -INFINITY;
  for (int i = lane_id; i < size; i += warpSize) {
    max_val = fmaxf(max_val, x[i]);
  }

  using WarpReduce = hipcub::WarpReduce<float>;
  __shared__ WarpReduce::TempStorage temp;
  __shared__ float shared_val;
  max_val = WarpReduce(temp).Reduce(max_val, hipcub::Max());

  if (threadIdx.x == 0) {
    shared_val = max_val;
  }
  __syncthreads();
  max_val = shared_val;

  float sum = 0.0f;
  for (int i = lane_id; i < size; i += warpSize) {
    sum += expf(x[i] - max_val);
  }

  sum = WarpReduce(temp).Sum(sum);
  if (threadIdx.x == 0) {
    shared_val = sum;
  }
  __syncthreads();
  sum = shared_val;
  for (int i = lane_id; i < size; i += warpSize) {
    y[i] = expf(x[i] - max_val) / sum;
  }
}

void softmax_inplace_kernel_cu(const tensor::Tensor& input, void* stream) {
  CHECK_EQ(input.is_empty(), false);
  CHECK(input.device_type() == base::DeviceType::kDeviceCUDA);
  int32_t size = static_cast<int32_t>(input.size());
  if (size < 1024) {
    constexpr int threads_num = 128;
    if (stream) {
      hipStream_t stream_ = static_cast<hipStream_t>(stream);
      row_softmax_fp32<<<1, threads_num, 0, stream_>>>(input.ptr<float>(), size);
    } else {
      row_softmax_fp32<<<1, threads_num>>>(const_cast<float*>(input.ptr<float>()), size);
    }
  } else {
    constexpr int threads_num = 1024;
    if (stream) {
      hipStream_t stream_ = static_cast<hipStream_t>(stream);
      row_softmax_fp32<<<1, threads_num, 0, stream_>>>(input.ptr<float>(), size);
    } else {
      row_softmax_fp32<<<1, threads_num>>>(input.ptr<float>(), size);
    }
  }
}
}  // namespace kernel