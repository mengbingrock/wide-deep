#include "hip/hip_runtime.h"

#include <cub/block/block_reduce.cuh>
#include <cub/warp/warp_reduce.cuh>
#include "rmsnorm_kernel.cuh"
namespace kernel {
__global__ void multihead_attention_fp32() {}

static __global__ void row_rmsnorm_f32(const float* in, const float* wei, float* out,
                                       const int size, const float eps) {
  const int tid = threadIdx.x;
  const int lane_id = tid % warpSize;

  float sum = 0.0f;
  for (int i = lane_id; i < size; i += warpSize) {
    sum += in[i] * in[i];
  }

  using WarpReduce = hipcub::WarpReduce<float, 32>;
  __shared__ typename WarpReduce::TempStorage temp;
  __shared__ float shared_val;
  sum = WarpReduce(temp).Reduce(sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    shared_val = sum;
  }
  __syncthreads();
  sum = shared_val;

  const float scale = rsqrtf(sum / static_cast<float>(size) + eps);
  for (int i = lane_id; i < size; i += warpSize) {
    out[i] = scale * in[i] * wei[i];
  }
}

void rmsnorm_kernel_cu(const tensor::Tensor& input, const tensor::Tensor& weight,
                       const tensor::Tensor& output, void* stream) {
  CHECK(!input.is_empty());
  CHECK(!weight.is_empty());
  CHECK(!output.is_empty());

  CHECK(input.device_type() == base::DeviceType::kDeviceCUDA &&
        weight.device_type() == base::DeviceType::kDeviceCUDA &&
        output.device_type() == base::DeviceType::kDeviceCUDA);

  const float eps = 1e-5f;
  const int32_t size = static_cast<int32_t>(input.size());
  const float* in_ptr = input.ptr<float>();
  const float* wei_ptr = weight.ptr<float>();
  float* out_ptr = const_cast<float*>(output.ptr<float>());
  if (size < 1024) {
    constexpr int threads_num = 128;
    if (stream) {
      hipStream_t stream_ = static_cast<hipStream_t>(stream);
      row_rmsnorm_f32<<<1, threads_num, 0, stream_>>>(in_ptr, wei_ptr, out_ptr, size, eps);
    } else {
      row_rmsnorm_f32<<<1, threads_num>>>(in_ptr, wei_ptr, out_ptr, size, eps);
    }
  } else {
    constexpr int threads_num = 1024;
    if (stream) {
      hipStream_t stream_ = static_cast<hipStream_t>(stream);
      row_rmsnorm_f32<<<1, threads_num, 0, stream_>>>(in_ptr, wei_ptr, out_ptr, size, eps);
    } else {
      row_rmsnorm_f32<<<1, threads_num>>>(in_ptr, wei_ptr, out_ptr, size, eps);
    }
  }
}
}  // namespace kernel
