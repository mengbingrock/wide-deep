#include "hip/hip_runtime.h"
#include "rope_kernel.cuh"
namespace kernel {

__device__ void rope_calc(float fcr, float fci, float* vec, int32_t idx) {
  float v0 = vec[idx];
  float v1 = vec[idx + 1];
  vec[idx] = v0 * fcr - v1 * fci;
  vec[idx + 1] = v0 * fci + v1 * fcr;
}

__global__ void rope_kernel_cu_fp32(int pos, int dim, int kv_dim, int head_size,
                                    const float* input_q, const float* input_k) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  idx = idx * 2;
  if (idx >= dim) {
    return;
  }

  int head_dim = idx % head_size;
  float freq = 1.0f / pow(10000.0f, static_cast<float>(head_dim) / static_cast<float>(head_size));
  float val = static_cast<float>(pos) * freq;
  float fcr = cosf(val);
  float fci = sinf(val);
  rope_calc(fcr, fci, const_cast<float*>(input_q), idx);
  if (idx >= kv_dim) {
    return;
  }
  rope_calc(fcr, fci, const_cast<float*>(input_k), idx);
}

void rope_kernel_cu(int32_t dim, int32_t kv_dim, int32_t head_size, const tensor::Tensor& input_q,
                    const tensor::Tensor& input_k, const tensor::Tensor& input_pos, void* stream) {
  const int32_t pos = *input_pos.ptr<int32_t>(0);
  int threads = 512;
  int blocks = (dim + threads - 1) / threads;
  if (stream) {
    hipStream_t stream_ = static_cast<hipStream_t>(stream);
    rope_kernel_cu_fp32<<<blocks, threads, 0, stream_>>>(
        pos, dim, kv_dim, head_size, input_q.ptr<float>(), input_k.ptr<float>());
  } else {
    rope_kernel_cu_fp32<<<blocks, threads>>>(pos, dim, kv_dim, head_size, input_q.ptr<float>(),
                                             input_k.ptr<float>());
  }
}
}  // namespace kernel